#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <math.h>
#include <assert.h>
#include <time.h>
#include <sys/time.h> 
#include "cudamat_conv_kernels.cuh"

using namespace std;


/*
 * images:   (numImgColors, imgPixels, numImages)
 * filters:   (numFilterColors, filterPixels, numFilters)
 * targets:   (numFilters, numModules, numImages)
 */

void filterActs(cudamat* images, cudamat* filters, cudamat* targets,
          int numModulesX, int paddingStart, int moduleStride,
          int numImgColors, int numGroups,
          float scaleTargets, float scaleOutput, bool conv) {
  int numFilterColors = numImgColors / numGroups;   
  int numFilters = filters->size[0];
  int numModules = numModulesX * numModulesX;
  int numImages = images->size[0];
  int imgPixels = images->size[1]/numImgColors;
  int imgSize = int(sqrt(imgPixels));
  int filterModuleMult = conv ? 1 : numModules;
  
  assert(numGroups > 1 || (numImgColors > 0 && (numImgColors <= 3 || numImgColors % 2 == 0)));
  assert(numGroups == 1 || numFilterColors % 2 == 0);
  assert(numFilters % (16 * numGroups) == 0);
  assert(numImgColors % numGroups == 0);
  assert(imgSize * imgSize == imgPixels);
  assert(images->size[1] == imgPixels * numImgColors);
  int numFiltersPerGroup = numFilters / numGroups;

  int imgStride = images->size[0]; //images.getStride(); // images does not need to be a contiguous matrix

  int filterPixels = filters->size[1] / (filterModuleMult * numFilterColors);
  int filterSize = int(sqrt(filterPixels));
  assert(filterSize * filterSize == filterPixels);

  assert(filters->size[1] == filterModuleMult * numFilterColors * filterPixels);

  // These routines don't handle the case when only part of the image is visited in the convolution
  assert(paddingStart <= 0 && paddingStart + (numModules-1)*moduleStride + filterSize >= imgSize);
  assert(moduleStride <= filterSize);
  
  /*
  assert(!images.isTrans());
  assert(!filters.isTrans());
  assert(!targets.isTrans());

  assert(filters.isContiguous());
  assert(targets.isContiguous());*/

  dim3 blocks = numFiltersPerGroup % 32 == 0 ? dim3(DIVUP(numImages, 32 * 4), (numModules * numFilters) / (4 * 8))
                        : dim3(DIVUP(numImages, 32 * 4), (numModules * numFilters) / (4 * 4));
  dim3 threads(32, 4);
  bool checkImgBounds = numImages % 128 != 0;
  //if (scaleTargets == 0) {
  //  targets.resize(numFilters * numModules, numImages);
  //} else {
  assert(targets->size[1] == numFilters * numModules);
  assert(targets->size[0] == numImages);
  //}
  

  if (numImgColors <= 3) {
    assert(numGroups == 1); // It has to be based on above definitions, but just to be sure.
    if (scaleTargets == 0) { // don't scale
      if (numImgColors == 1) {
        if (checkImgBounds) {
          if (numFilters % 32 == 0) {
		   // WTF is this shit? Why does it set everything to zero? 
		   // There has got to be an explanation.
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 1, false, true >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 8, 1, false, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
			 numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 1, false, true >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 4, 1, false, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          }
        } else {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 1, false, false >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 8, 1, false, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 1, false, false >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 4, 1, false, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          }
        }
      } else if (numImgColors == 2) {
        if (checkImgBounds) {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 2, false, true >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 8, 2, false, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 2, false, true >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 4, 2, false, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          }
        } else {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 2, false, false >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 8, 2, false, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 2, false, false >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 4, 2, false, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          }
        }
      } else if (numImgColors == 3) {
        if (checkImgBounds) {
           if (numFilters % 32 == 0) {
             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 3, false, true >, hipFuncCachePreferShared);
             filterActs_YxX_color < 4, 32, 4, 8, 3, false, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                   numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
           } else {
             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 3, false, true >, hipFuncCachePreferShared);
             filterActs_YxX_color < 4, 32, 4, 4, 3, false, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                   numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
           }
        } else {
           if (numFilters % 32 == 0) {
             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 3, false, false >, hipFuncCachePreferShared);
             filterActs_YxX_color < 4, 32, 4, 8, 3, false, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                   numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
           } else {
             hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 3, false, false >, hipFuncCachePreferShared);
             filterActs_YxX_color < 4, 32, 4, 4, 3, false, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                   numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
           }
        }
      }
    } else { // do scale
      if (numImgColors == 1) {
        if (checkImgBounds) {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 1, true, true >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 8, 1, true, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 1, true, true >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 4, 1, true, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          }
        } else {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 1, true, false >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 8, 1, true, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 1, true, false >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 4, 1, true, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          }
        }
      } else if (numImgColors == 2) {
        if (checkImgBounds) {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 2, true, true >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 8, 2, true, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 2, true, true >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 4, 2, true, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          }
        } else {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 2, true, false >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 8, 2, true, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 2, true, false >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 4, 2, true, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          }
        }
      } else if (numImgColors == 3) {
        if (checkImgBounds) {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 3, true, true >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 8, 3, true, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 3, true, true >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 4, 3, true, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          }
        } else {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 3, true, false >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 8, 3, true, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 3, true, false >, hipFuncCachePreferShared);
            filterActs_YxX_color < 4, 32, 4, 4, 3, true, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                  numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput, conv);
          }
        }
      }
    }
  } else {
    if (scaleTargets == 0) { // don't scale
      if (checkImgBounds) {
        if (numFiltersPerGroup % 32 == 0) {
          hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 8, 2, false, true >, hipFuncCachePreferShared);
          filterActs_YxX_sparse < 4, 32, 4, 8, 2, false, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
        } else {
          hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 4, 2, false, true >, hipFuncCachePreferShared);
          filterActs_YxX_sparse < 4, 32, 4, 4, 2, false, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
        }
      } else {
        if (numFiltersPerGroup % 32 == 0) {
          hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 8, 2, false, false >, hipFuncCachePreferShared);
          filterActs_YxX_sparse < 4, 32, 4, 8, 2, false, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
        } else {
          hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 4, 2, false, false >, hipFuncCachePreferShared);
          filterActs_YxX_sparse < 4, 32, 4, 4, 2, false, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
        }
      }
    } else { // do scale
      if (checkImgBounds) {
        if (numFiltersPerGroup % 32 == 0) {
          hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 8, 2, false, true >, hipFuncCachePreferShared);
          filterActs_YxX_sparse < 4, 32, 4, 8, 2, true, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
        } else {
          hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 4, 2, false, true >, hipFuncCachePreferShared);
          filterActs_YxX_sparse < 4, 32, 4, 4, 2, true, true > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
        }
      } else {
        if (numFiltersPerGroup % 32 == 0) {
          hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 8, 2, false, false >, hipFuncCachePreferShared);
          filterActs_YxX_sparse < 4, 32, 4, 8, 2, true, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
        } else {
          hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 4, 2, false, false >, hipFuncCachePreferShared);
          filterActs_YxX_sparse < 4, 32, 4, 4, 2, true, false > <<<blocks, threads>>>(images->data_device, filters->data_device, targets->data_device,
                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput, conv);
        }
      }
    }
  }
  
  getLastCudaError("filterActs: kernel execution failed");
}

/*
 * hidActs:   (numFilters, numModules, numImages)
 * filters:   (numFilterColors, filterPixels, numFilters)        if conv
 *       (numModules, numFilterColors, filterPixels, numFilters)  otherwise
 * targets:   (numImageColors, imgPixels, numImages)
 */
void imgActs(cudamat* hidActs, cudamat* filters, cudamat* targets,
       int imgSize, int paddingStart, int moduleStride, int numImgColors, int numGroups,
       float scaleTargets, float scaleOutput, bool conv) {
  int numFilterColors = numImgColors / numGroups;
  int numImages = hidActs->size[0];
  int numFilters = filters->size[0];
  //int numFiltersPerGroup = numFilters / numGroups;
  int numModules = hidActs->size[1] / numFilters;
  int filterModuleMult = conv ? 1 : numModules;
  int filterPixels = filters->size[1] / (filterModuleMult * numFilterColors);
  int filterSize = sqrt(filterPixels);
  int imgPixels = imgSize * imgSize;
  int numModulesX = sqrt(numModules);
  
  assert(numImgColors % numGroups == 0);
  assert(numFilters % (16*numGroups) == 0);
  assert(numGroups > 1 || (numImgColors > 0 && (numImgColors <= 3 || numImgColors % 2 == 0)));
  assert(numGroups == 1 || numFilterColors % 4 == 0);

  assert(filterPixels == filterSize * filterSize);
  assert(hidActs->size[1] == numModules * numFilters);
  assert(filters->size[1] == filterModuleMult * numFilterColors * filterPixels);
  assert(numModules == numModulesX * numModulesX);

  /*
  assert(hidActs.isContiguous());
  assert(filters.isContiguous());

  assert(!hidActs.isTrans());
  assert(!filters.isTrans());
  assert(!targets.isTrans());*/
  // These routines don't handle the case when only part of the image is visited in the convolution
  assert(paddingStart <= 0 && paddingStart + (numModules-1)*moduleStride + filterSize >= imgSize);
  assert(moduleStride <= filterSize);
  
  //assert(targets.isContiguous()); // no stride support here!

  dim3 blocks;
  dim3 threads(16,16);
  int colorsPerThread;
  bool checkCaseBounds;
  if (numFilterColors % 8 == 0) {
    threads = dim3(32, 4);
    colorsPerThread = numFilterColors % 16 == 0 ? 4 : 2;
    int imgsPerThread = 4;
    assert(numFilterColors % (threads.y * colorsPerThread) == 0);
    checkCaseBounds = numImages % (threads.x * imgsPerThread) != 0;
    blocks = dim3(DIVUP(numImages, threads.x*imgsPerThread) * (numImgColors/(threads.y*colorsPerThread)), imgPixels);
  } else if (numFilterColors > 3) {
    colorsPerThread = numFilterColors % 4 == 0 ? 4 : 2;
    blocks = dim3(DIVUP(numImages,16*8) * (numImgColors / colorsPerThread), DIVUP(imgSize,4) * DIVUP(imgSize,4));
    checkCaseBounds = numImages % (16*8) != 0;
  } else {
    blocks = dim3(DIVUP(numImages,16*8), DIVUP(imgSize,4) * DIVUP(imgSize,4));
    checkCaseBounds = numImages % (16*8) != 0;
  }
  
  //if (scaleTargets == 0) { // do not scale or use targets matrix
  //  targets.resize(numImgColors*imgPixels, numImages);
  //} else {
  assert(targets->size[1] == numImgColors * imgPixels);
  assert(targets->size[0] == numImages);
  //}
  
  if (conv) { // convolutional units
    if (scaleTargets == 0) { // do not scale or use targets matrix
      if (numFilterColors % 8 == 0) {
        if (checkCaseBounds) {
          if (numFilterColors % 16 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 4, false, true, true>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 4, false, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 2, false, true, true>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 2, false, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilterColors % 16 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 4, false, false, true>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 4, false, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 2, false, false, true>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 2, false, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        }
      } else if (numFilterColors > 3) {
        if (checkCaseBounds) {
          if (colorsPerThread == 4) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 4, false, true, true>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 4, false, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 2, false, true, true>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 2, false, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        } else {
          if (colorsPerThread == 4) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 4, false, false, true>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 4, false, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 2, false, false, true>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 2, false, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        }
      } else {
        if (checkCaseBounds) {
          if (numFilterColors == 1) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 1, false, true, true>, hipFuncCachePreferShared);
            img_acts_color<8, 1, false, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 2) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 2, false, true, true>, hipFuncCachePreferShared);
            img_acts_color<8, 2, false, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 3) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 3, false, true, true>, hipFuncCachePreferShared);
            img_acts_color<8, 3, false, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilterColors == 1) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 1, false, false, true>, hipFuncCachePreferShared);
            img_acts_color<8, 1, false, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 2) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 2, false, false, true>, hipFuncCachePreferShared);
            img_acts_color<8, 2, false, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 3) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 3, false, false, true>, hipFuncCachePreferShared);
            img_acts_color<8, 3, false, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          }
        }
      }
    } else { // do scale
      if (numFilterColors % 8 == 0) {
        if (checkCaseBounds) {
          if (numFilterColors % 16 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 4, true, true, true>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 4, true, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 2, true, true, true>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 2, true, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilterColors % 16 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 4, true, false, true>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 4, true, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 2, true, false, true>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 2, true, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        }
      } else if (numFilterColors > 3) {
        if (checkCaseBounds) {
          if (colorsPerThread == 4) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 4, true, true, true>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 4, true, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 2, true, true, true>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 2, true, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        } else {
          if (colorsPerThread == 4) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 4, true, false, true>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 4, true, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 2, true, false, true>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 2, true, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        }
      } else {
        if (checkCaseBounds) {
          if (numFilterColors == 1) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 1, true, true, true>, hipFuncCachePreferShared);
            img_acts_color<8, 1, true, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 2) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 2, true, true, true>, hipFuncCachePreferShared);
            img_acts_color<8, 2, true, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 3) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 3, true, true, true>, hipFuncCachePreferShared);
            img_acts_color<8, 3, true, true, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilterColors == 1) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 1, true, false, true>, hipFuncCachePreferShared);
            img_acts_color<8, 1, true, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 2) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 2, true, false, true>, hipFuncCachePreferShared);
            img_acts_color<8, 2, true, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 3) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 3, true, false, true>, hipFuncCachePreferShared);
            img_acts_color<8, 3, true, false, true><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          }
        }
      }
    }
  } else { // local, unshared units
    if (scaleTargets == 0) { // do not scale or use targets matrix
      if (numFilterColors % 8 == 0) {
        if (checkCaseBounds) {
          if (numFilterColors % 16 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 4, false, true, false>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 4, false, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 2, false, true, false>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 2, false, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilterColors % 16 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 4, false, false, false>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 4, false, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 2, false, false, false>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 2, false, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        }
      } else if (numFilterColors > 3) {
        if (checkCaseBounds) {
          if (colorsPerThread == 4) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 4, false, true, false>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 4, false, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 2, false, true, false>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 2, false, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        } else {
          if (colorsPerThread == 4) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 4, false, false, false>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 4, false, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 2, false, false, false>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 2, false, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        }
      } else {
        if (checkCaseBounds) {
          if (numFilterColors == 1) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 1, false, true, false>, hipFuncCachePreferShared);
            img_acts_color<8, 1, false, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 2) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 2, false, true, false>, hipFuncCachePreferShared);
            img_acts_color<8, 2, false, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 3) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 3, false, true, false>, hipFuncCachePreferShared);
            img_acts_color<8, 3, false, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilterColors == 1) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 1, false, false, false>, hipFuncCachePreferShared);
            img_acts_color<8, 1, false, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 2) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 2, false, false, false>, hipFuncCachePreferShared);
            img_acts_color<8, 2, false, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 3) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 3, false, false, false>, hipFuncCachePreferShared);
            img_acts_color<8, 3, false, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          }
        }
      }
    } else { // do scale
      if (numFilterColors % 8 == 0) {
        if (checkCaseBounds) {
          if (numFilterColors % 16 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 4, true, true, false>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 4, true, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 2, true, true, false>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 2, true, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilterColors % 16 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 4, true, false, false>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 4, true, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_img_acts_manycolor<4), 32, 4, 2, true, false, false>, hipFuncCachePreferShared);
            conv_img_acts_manycolor<4, 32, 4, 2, true, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        }
      } else if (numFilterColors > 3) {
        if (checkCaseBounds) {
          if (colorsPerThread == 4) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 4, true, true, false>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 4, true, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 2, true, true, false>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 2, true, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        } else {
          if (colorsPerThread == 4) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 4, true, false, false>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 4, true, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_mediumcolor<8), 2, true, false, false>, hipFuncCachePreferShared);
            img_acts_mediumcolor<8, 2, true, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, numImgColors, numGroups, scaleTargets, scaleOutput);
          }
        }
      } else {
        if (checkCaseBounds) {
          if (numFilterColors == 1) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 1, true, true, false>, hipFuncCachePreferShared);
            img_acts_color<8, 1, true, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 2) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 2, true, true, false>, hipFuncCachePreferShared);
            img_acts_color<8, 2, true, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 3) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 3, true, true, false>, hipFuncCachePreferShared);
            img_acts_color<8, 3, true, true, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilterColors == 1) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 1, true, false, false>, hipFuncCachePreferShared);
            img_acts_color<8, 1, true, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 2) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 2, true, false, false>, hipFuncCachePreferShared);
            img_acts_color<8, 2, true, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          } else if (numFilterColors == 3) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(img_acts_color<8), 3, true, false, false>, hipFuncCachePreferShared);
            img_acts_color<8, 3, true, false, false><<<blocks, threads>>>(hidActs->data_device, filters->data_device, targets->data_device,
                              numModulesX, numImages, numFilters, filterSize, imgSize, paddingStart, moduleStride, scaleTargets, scaleOutput);
          }
        }
      }
    }
  }
  
  getLastCudaError("imgActs: kernel execution failed");
}

void weightActs(cudamat* images, cudamat* hidActs, cudamat* targets,
    int numModulesX, int filterSize, int paddingStart, int moduleStride, int numImgColors,
    int numGroups, int partialSum, float scaleTargets, float scaleOutput) {
  int numFilterColors = numImgColors / numGroups;
  int imgStride = images->size[0];
  int numImages = images->size[0];
  int imgPixels = images->size[1] / numImgColors;
  int imgSize = int(sqrt(imgPixels));
  int numModules = numModulesX * numModulesX;
  int numFilters = hidActs->size[1] / numModules;
  int numFiltersPerGroup = numFilters / numGroups;
  
  assert(numImgColors % numGroups == 0);
  assert(numFilters % (16*numGroups) == 0);
  assert(numGroups > 1 || (numImgColors > 0 && (numImgColors <= 3 || numImgColors % 4 == 0)));
  assert(numGroups == 1 || numFilterColors % 4 == 0);
  assert(imgSize * imgSize == imgPixels);
  assert(images->size[1] == imgPixels * numImgColors);

  int filterPixels = filterSize * filterSize;
  partialSum = partialSum == 0 ? numModules : partialSum;

  assert(numModules % partialSum == 0);
  assert(hidActs->size[0] == numImages);

  // These routines don't handle the case when only part of the image is visited in the convolution
  assert(paddingStart <= 0 && paddingStart + (numModules-1)*moduleStride + filterSize >= imgSize);
  assert(moduleStride <= filterSize);
  
  assert(numModules * numFilters == hidActs->size[1]);

  /*
  assert(!images.isTrans());
  assert(!hidActs.isTrans());
  assert(hidActs.isContiguous());

  assert(!targets.isTrans());
  assert(targets.isContiguous());*/
  
  int preloadCases = 32;

  dim3 blocks, threads;
  int bx, by;
  int pixelsPerThread, filtersPerThread, colorsPerThread;
  // Worth playing with these parameters to find best values for your problem.
  // These values work relatively well, but not optimal for all problems.
  if (numFilterColors > 3) {
    filtersPerThread = numFiltersPerGroup % 32 == 0 ? 2 : 1;
    colorsPerThread = numFilterColors % 8 == 0 ? 8 : 4;
    by = numFiltersPerGroup % 64 == 0 ? 4 : 8;
    bx = numFiltersPerGroup % 64 == 0 ? 32 : 16;
    blocks = dim3((numModules/partialSum)*(numFilters/(bx*filtersPerThread)), DIVUP(filterPixels, by) * (numFilterColors / colorsPerThread));
  } else {
    assert(numGroups == 1); // Just for sanity
    pixelsPerThread = numFilters % 32 == 0 ? (numImgColors == 1 ? 8 : 5) : (numImgColors == 1 ? 5 : 2);
    by = numFilters % 32 == 0 ? 4 : 8; // by == 4 seems to work best
    bx = numFilters % 32 == 0 ? 32 : 16; 
    blocks = dim3((numModules/partialSum)*(numFilters/bx), DIVUP(filterPixels, by*pixelsPerThread));
  }
  assert((by * bx) % preloadCases == 0);
  threads = dim3(bx, by);
  bool checkCaseBounds = numImages % 32 != 0;
  
  //if (scaleTargets == 0) {
  //  targets.resize((numModules/partialSum) * numFilterColors*filterPixels, numFilters);
  //} else {
  // printf("targets-%d-%d-%d-%d-%d-%d-%d\n",targets->size[1],targets->size[0],numModules,partialSum,numFilterColors,filterPixels,numFilters);
  // printf("salam-%d\n",(numModules/partialSum) * numFilterColors*filterPixels);
  assert(targets->size[1] == (numModules/partialSum) * numFilterColors*filterPixels);
  assert(targets->size[0] == numFilters);
  //}
  if (numFilterColors > 3) {
    if (scaleTargets == 0) { // do not scale
      if (numFiltersPerGroup % 64 == 0) {
        if (numFilterColors % 8 == 0) {
          if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<4),32,2,8,32, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<4,32,2,8,32,false, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<4),32,2,8,32, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<4,32,2,8,32,false, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          }
        } else {
          if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<4),32,2,4,32, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<4,32,2,4,32,false, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<4),32,2,4,32, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<4,32,2,4,32,false, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          }
        }
      } else if (numFiltersPerGroup % 32 == 0) {
        if (numFilterColors % 8 == 0) {
          if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,2,8,32, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,2,8,32,false, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,2,8,32, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,2,8,32,false, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          }
        } else {
          if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,2,4,32, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,2,4,32,false, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,2,4,32, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,2,4,32,false, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          }
        }
      } else {
        if (numFilterColors % 8 == 0) {
          if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,1,8,32, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,1,8,32,false, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,1,8,32, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,1,8,32,false, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          }
        } else {
          if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,1,4,32, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,1,4,32,false, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,1,4,32, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,1,4,32,false, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          }
        }
      }
    } else {

      if (numFiltersPerGroup % 64 == 0) {
        if (numFilterColors % 8 == 0) {
          if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<4),32,2,8,32, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<4,32,2,8,32,true, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<4),32,2,8,32, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<4,32,2,8,32,true, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          }
        } else {
          if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<4),32,2,4,32, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<4,32,2,4,32,true, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<4),32,2,4,32, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<4,32,2,4,32,true, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          }
        }
      } else if (numFiltersPerGroup % 32 == 0) {
        if (numFilterColors % 8 == 0) {
          if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,2,8,32, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,2,8,32,true, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,2,8,32, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,2,8,32,true, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          }
        } else {
          if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,2,4,32, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,2,4,32,true, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,2,4,32, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,2,4,32,true, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          }
        }
      } else {
        if (numFilterColors % 8 == 0) {
          if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,1,8,32, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,1,8,32,true, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,1,8,32, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,1,8,32,true, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          }
        } else {
          if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,1,4,32, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,1,4,32,true, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_mc_mf<8),16,1,4,32, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_mc_mf<8,16,1,4,32,true, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                            numImages, numFilters, numModulesX, imgSize, filterSize,
                                            paddingStart, moduleStride, imgStride, numImgColors, numGroups, partialSum, scaleTargets, scaleOutput);
          }
        }
      }
    }
  } else { // numColors in 1,2,3
    if (scaleTargets == 0) { // do not scale
      if (numFilterColors == 1) {
        if (checkCaseBounds) {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<4),32,8,32,1, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_c<4,32,8,32,1,false, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<8),16,5,32,1, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_c<8,16,5,32,1,false, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<4),32,8,32,1, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_c<4,32,8,32,1,false, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<8),16,5,32,1, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_c<8,16,5,32,1,false, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          }
        }
      } else if (numFilterColors == 2) {
        if (checkCaseBounds) {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<4),32,5,32,2, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_c<4,32,5,32,2,false, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<8),16,2,32,2, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_c<8,16,2,32,2,false, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<4),32,5,32,2, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_c<4,32,5,32,2,false, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<8),16,2,32,2, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_c<8,16,2,32,2,false, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          }
        }
      } else if (numFilterColors == 3) {
        if (checkCaseBounds) {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<4),32,5,32,3, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_c<4,32,5,32,3,false, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<8),16,2,32,3, false, true>, hipFuncCachePreferShared);
            conv_weight_acts_c<8,16,2,32,3,false, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<4),32,5,32,3, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_c<4,32,5,32,3,false, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<8),16,2,32,3, false, false>, hipFuncCachePreferShared);
            conv_weight_acts_c<8,16,2,32,3,false, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          }
        }
      }

    } else { // do scale
      if (numFilterColors == 1) {
        if (checkCaseBounds) {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<4),32,8,32,1, true, true>, hipFuncCachePreferShared);
            conv_weight_acts_c<4,32,8,32,1,true, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<8),16,5,32,1, true, true>, hipFuncCachePreferShared);
            conv_weight_acts_c<8,16,5,32,1,true, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<4),32,8,32,1, true, false>, hipFuncCachePreferShared);
            conv_weight_acts_c<4,32,8,32,1,true, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<8),16,5,32,1, true, false>, hipFuncCachePreferShared);
            conv_weight_acts_c<8,16,5,32,1,true, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          }
        }
      } else if (numFilterColors == 2) {
        if (checkCaseBounds) {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<4),32,5,32,2, true, true>, hipFuncCachePreferShared);
            conv_weight_acts_c<4,32,5,32,2,true, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<8),16,2,32,2, true, true>, hipFuncCachePreferShared);
            conv_weight_acts_c<8,16,2,32,2,true, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<4),32,5,32,2, true, false>, hipFuncCachePreferShared);
            conv_weight_acts_c<4,32,5,32,2,true, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<8),16,2,32,2, true, false>, hipFuncCachePreferShared);
            conv_weight_acts_c<8,16,2,32,2,true, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          }
        }
      } else if (numFilterColors == 3) {
        if (checkCaseBounds) {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<4),32,5,32,3, true, true>, hipFuncCachePreferShared);
            conv_weight_acts_c<4,32,5,32,3,true, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<8),16,2,32,3, true, true>, hipFuncCachePreferShared);
            conv_weight_acts_c<8,16,2,32,3,true, true><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          }
        } else {
          if (numFilters % 32 == 0) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<4),32,5,32,3, true, false>, hipFuncCachePreferShared);
            conv_weight_acts_c<4,32,5,32,3,true, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(conv_weight_acts_c<8),16,2,32,3, true, false>, hipFuncCachePreferShared);
            conv_weight_acts_c<8,16,2,32,3,true, false><<<blocks, threads>>>(images->data_device, hidActs->data_device, targets->data_device,
                                numImages, numFilters, numModulesX, imgSize, filterSize, paddingStart, moduleStride, imgStride, partialSum, scaleTargets, scaleOutput);
          }
        }
      }
    }
  }
  getLastCudaError("weightActs: kernel execution failed");
}

/*
 * imgs:    (numFilters, imgPixels, numImages)
 * target:   (numFilters, outputs, numImages)
 */
template<class Pooler>
void convLocalPool(cudamat* images, cudamat* target, int numFilters, int subsX, int startX, int strideX, int outputsX, Pooler pooler) {
  int numImages = images->size[0];
  int imgPixels = images->size[1] / numFilters;
  assert(images->size[1] == numFilters * imgPixels);
  int imgSize = int(sqrt(imgPixels));
  assert(imgSize * imgSize == imgPixels);
  
  /*
  assert(!images.isTrans());
  assert(!target.isTrans());
  assert(images.isContiguous());
  */
//  assert(numFilters % 4 == 0);
//  assert(numImages % 128 == 0);
  
  // int outputs = outputsX * outputsX;
  //target.resize(numFilters*outputs, numImages);

  if (strideX == 1 && subsX >= 6) {
    int imgsPerThread = 8;
    int filtersPerThread = numFilters % 4 == 0 ? 4 : numFilters % 3 == 0 ? 3 : numFilters % 2 == 0 ? 2 : 1;
    int bx = 8;
    bool checkCaseBounds = numImages % (bx*imgsPerThread) != 0;
    assert((imgsPerThread * bx) % 32 == 0);
    assert(numFilters % filtersPerThread == 0);
    dim3 threads(bx, 16);
    dim3 blocks(DIVUP(outputsX, 4) * DIVUP(numImages, bx*imgsPerThread), DIVUP(outputsX, 4) * numFilters / filtersPerThread);
    if (filtersPerThread == 1) {
       if (checkCaseBounds) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalPool2<Pooler), 8, 8, 1, true>, hipFuncCachePreferShared);
        kLocalPool2<Pooler, 8, 8, 1, true><<<blocks, threads>>>(images->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalPool2<Pooler), 8, 8, 1, false>, hipFuncCachePreferShared);
        kLocalPool2<Pooler, 8, 8, 1, false><<<blocks, threads>>>(images->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      }
    } else if (filtersPerThread == 2) {
      if (checkCaseBounds) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalPool2<Pooler), 8, 8, 2, true>, hipFuncCachePreferShared);
        kLocalPool2<Pooler, 8, 8, 2, true><<<blocks, threads>>>(images->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalPool2<Pooler), 8, 8, 2, false>, hipFuncCachePreferShared);
        kLocalPool2<Pooler, 8, 8, 2, false><<<blocks, threads>>>(images->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      }
    } else if (filtersPerThread == 3) {
      if (checkCaseBounds) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalPool2<Pooler), 8, 8, 3, true>, hipFuncCachePreferShared);
        kLocalPool2<Pooler, 8, 8, 3, true><<<blocks, threads>>>(images->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalPool2<Pooler), 8, 8, 3, false>, hipFuncCachePreferShared);
        kLocalPool2<Pooler, 8, 8, 3, false><<<blocks, threads>>>(images->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      }
    } else if (filtersPerThread == 4) {
      if (checkCaseBounds) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalPool2<Pooler), 8, 8, 4, true>, hipFuncCachePreferShared);
        kLocalPool2<Pooler, 8, 8, 4, true><<<blocks, threads>>>(images->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalPool2<Pooler), 8, 8, 4, false>, hipFuncCachePreferShared);
        kLocalPool2<Pooler, 8, 8, 4, false><<<blocks, threads>>>(images->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      }
    }
  } else {
    bool checkCaseBounds = numImages % 128 != 0;
    int filtersPerThread = numFilters % 8 == 0 ? 2 : 1;
    dim3 threads(32, 4);
    dim3 blocks(DIVUP(numImages,32*4) * outputsX, DIVUP(numFilters, 4 * filtersPerThread) * outputsX);
    if (filtersPerThread == 1) {
      if (checkCaseBounds) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalPool<Pooler), 4, 32, 4, 1, true>, hipFuncCachePreferL1);
        kLocalPool<Pooler, 4, 32, 4, 1, true><<<blocks, threads>>>(images->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, pooler);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalPool<Pooler), 4, 32, 4, 1, false>, hipFuncCachePreferL1);
        kLocalPool<Pooler, 4, 32, 4, 1, false><<<blocks, threads>>>(images->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, pooler);
      }
    } else {
      if (checkCaseBounds) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalPool<Pooler), 4, 32, 4, 2, true>, hipFuncCachePreferL1);
        kLocalPool<Pooler, 4, 32, 4, 2, true><<<blocks, threads>>>(images->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, pooler);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalPool<Pooler), 4, 32, 4, 2, false>, hipFuncCachePreferL1);
        kLocalPool<Pooler, 4, 32, 4, 2, false><<<blocks, threads>>>(images->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, pooler);
      }
    }

  }

  getLastCudaError("convLocalPool: kernel execution failed");
}

/*
 * imgs:    (numFilters, imgPixels, numImages)
 * maxGrads:  (numFilters, numOutputs, numImages)
 * rMaxActs:  (numFilters, numOutputs, numImages)
 * target:   (numFilters, imgPixels, numImages)
 */


void convLocalAvgUndo(cudamat* avgGrads, cudamat* target,int subsX, int startX, int strideX, int outputsX, float scaleTargets, float scaleOutput) {
  int outputs = outputsX * outputsX;
  int numImages = target->size[0];
  int numFilters = avgGrads->size[1] / outputs;
  int imgPixels = target->size[1] / numFilters;
  assert(target->size[1] == numFilters * imgPixels);
  int imgSize = int(sqrt(imgPixels));
  
  assert(imgSize * imgSize == imgPixels);
  assert(avgGrads->size[1] == numFilters * outputs);
  assert(avgGrads->size[0] == numImages);

  /*
  assert(!images.isTrans());
  assert(!target.isTrans());
  assert(!maxGrads.isTrans());
  assert(!maxActs.isTrans());
  assert(images.isContiguous());
  assert(maxGrads.isContiguous());
  assert(maxActs.isContiguous());
  assert(maxGrads.isSameDims(maxActs));
  */

  assert(numFilters % 16 == 0);
//  assert(numImages % 128 == 0);
  
  assert(strideX <= subsX);
  
  //target.resize(images);
  
  int checkCaseBounds = numImages % 128 != 0;
  dim3 threads(32, 4);
  dim3 blocks(DIVUP(numImages,32*4) * imgSize, (numFilters / (4 * 2)) * imgSize);
  
  if (checkCaseBounds) {
    if (scaleTargets == 0 && scaleOutput == 1) {
      kLocalAvgUndo<4, 32, 4, 2, false, true><<<blocks, threads>>>(avgGrads->data_device, target->data_device,
                              imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
    } else {
      kLocalAvgUndo<4, 32, 4, 2, true, true><<<blocks, threads>>>(avgGrads->data_device, target->data_device,
                              imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
    }
  } else {
    if (scaleTargets == 0 && scaleOutput == 1) {
      kLocalAvgUndo<4, 32, 4, 2, false, false><<<blocks, threads>>>(avgGrads->data_device, target->data_device,
                              imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
    } else {
      kLocalAvgUndo<4, 32, 4, 2, true, false><<<blocks, threads>>>(avgGrads->data_device, target->data_device,
                              imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
    }
  }

  getLastCudaError("convLocalMaxUndo: kernel execution failed");
}


void convLocalMaxUndo(cudamat* images, cudamat* maxGrads, cudamat* maxActs, cudamat* target,
           int subsX, int startX, int strideX, int outputsX, float scaleTargets, float scaleOutput) {
  int outputs = outputsX * outputsX;
  int numImages = images->size[0];
  int numFilters = maxGrads->size[1] / outputs;
  int imgPixels = images->size[1] / numFilters;
  assert(images->size[1] == numFilters * imgPixels);
  int imgSize = int(sqrt(imgPixels));
  
  assert(imgSize * imgSize == imgPixels);
  assert(maxGrads->size[1] == numFilters * outputs);
  assert(maxGrads->size[0] == numImages);

  /*
  assert(!images.isTrans());
  assert(!target.isTrans());
  assert(!maxGrads.isTrans());
  assert(!maxActs.isTrans());
  assert(images.isContiguous());
  assert(maxGrads.isContiguous());
  assert(maxActs.isContiguous());
  assert(maxGrads.isSameDims(maxActs));
  */

  assert(numFilters % 16 == 0);
//  assert(numImages % 128 == 0);
  
  assert(strideX <= subsX);
  
  //target.resize(images);
  
  int checkCaseBounds = numImages % 128 != 0;
  dim3 threads(32, 4);
  dim3 blocks(DIVUP(numImages,32*4) * imgSize, (numFilters / (4 * 2)) * imgSize);
  
  if (checkCaseBounds) {
    if (scaleTargets == 0 && scaleOutput == 1) {
      kLocalMaxUndo<4, 32, 4, 2, false, true><<<blocks, threads>>>(images->data_device, maxGrads->data_device, maxActs->data_device, target->data_device,
                              imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
    } else {
      kLocalMaxUndo<4, 32, 4, 2, true, true><<<blocks, threads>>>(images->data_device, maxGrads->data_device, maxActs->data_device, target->data_device,
                              imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
    }
  } else {
    if (scaleTargets == 0 && scaleOutput == 1) {
      kLocalMaxUndo<4, 32, 4, 2, false, false><<<blocks, threads>>>(images->data_device, maxGrads->data_device, maxActs->data_device, target->data_device,
                              imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
    } else {
      kLocalMaxUndo<4, 32, 4, 2, true, false><<<blocks, threads>>>(images->data_device, maxGrads->data_device, maxActs->data_device, target->data_device,
                              imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, scaleTargets, scaleOutput);
    }
  }

  getLastCudaError("convLocalMaxUndo: kernel execution failed");
}

/*
 * imgs:    (numFilters, imgPixels, numImages)
 * rnd:    (numFilters, imgPixels, numImages)
 * target:   (numFilters, outputs, numImages)
 */
template<class Pooler>
void convLocalProbPool(cudamat* images, cudamat* rnd, cudamat* target, int numFilters, int subsX, int startX, int strideX, int outputsX, Pooler pooler) {
  int numImages = images->size[0];
  int imgPixels = images->size[1] / numFilters;
  assert(images->size[1] == numFilters * imgPixels);
  int imgSize = int(sqrt(imgPixels));
  assert(imgSize * imgSize == imgPixels);
  assert(rnd->size[0] == images->size[0]);
  assert(rnd->size[1] == images->size[1]);
  /*
  assert(!images.isTrans());
  assert(!target.isTrans());
  assert(images.isContiguous());
  */
//  assert(numFilters % 4 == 0);
//  assert(numImages % 128 == 0);
  
  // int outputs = outputsX * outputsX;
  //target.resize(numFilters*outputs, numImages);

  if (strideX == 1 && subsX >= 6) {
    int imgsPerThread = 8;
    int filtersPerThread = numFilters % 4 == 0 ? 4 : numFilters % 3 == 0 ? 3 : numFilters % 2 == 0 ? 2 : 1;
    int bx = 8;
    bool checkCaseBounds = numImages % (bx*imgsPerThread) != 0;
    assert((imgsPerThread * bx) % 32 == 0);
    assert(numFilters % filtersPerThread == 0);
    dim3 threads(bx, 16);
    dim3 blocks(DIVUP(outputsX, 4) * DIVUP(numImages, bx*imgsPerThread), DIVUP(outputsX, 4) * numFilters / filtersPerThread);
    if (filtersPerThread == 1) {
       if (checkCaseBounds) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalProbPool2<Pooler), 8, 8, 1, true>, hipFuncCachePreferShared);
        kLocalProbPool2<Pooler, 8, 8, 1, true><<<blocks, threads>>>(images->data_device, rnd->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalProbPool2<Pooler), 8, 8, 1, false>, hipFuncCachePreferShared);
        kLocalProbPool2<Pooler, 8, 8, 1, false><<<blocks, threads>>>(images->data_device, rnd->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      }
    } else if (filtersPerThread == 2) {
      if (checkCaseBounds) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalProbPool2<Pooler), 8, 8, 2, true>, hipFuncCachePreferShared);
        kLocalProbPool2<Pooler, 8, 8, 2, true><<<blocks, threads>>>(images->data_device, rnd->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalProbPool2<Pooler), 8, 8, 2, false>, hipFuncCachePreferShared);
        kLocalProbPool2<Pooler, 8, 8, 2, false><<<blocks, threads>>>(images->data_device, rnd->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      }
    } else if (filtersPerThread == 3) {
      if (checkCaseBounds) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalProbPool2<Pooler), 8, 8, 3, true>, hipFuncCachePreferShared);
        kLocalProbPool2<Pooler, 8, 8, 3, true><<<blocks, threads>>>(images->data_device, rnd->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalProbPool2<Pooler), 8, 8, 3, false>, hipFuncCachePreferShared);
        kLocalProbPool2<Pooler, 8, 8, 3, false><<<blocks, threads>>>(images->data_device, rnd->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      }
    } else if (filtersPerThread == 4) {
      if (checkCaseBounds) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalProbPool2<Pooler), 8, 8, 4, true>, hipFuncCachePreferShared);
        kLocalProbPool2<Pooler, 8, 8, 4, true><<<blocks, threads>>>(images->data_device, rnd->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalProbPool2<Pooler), 8, 8, 4, false>, hipFuncCachePreferShared);
        kLocalProbPool2<Pooler, 8, 8, 4, false><<<blocks, threads>>>(images->data_device, rnd->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, outputsX, pooler);
      }
    }
  } else {
    bool checkCaseBounds = numImages % 128 != 0;
    int filtersPerThread = numFilters % 8 == 0 ? 2 : 1;
    dim3 threads(32, 4);
    dim3 blocks(DIVUP(numImages,32*4) * outputsX, DIVUP(numFilters, 4 * filtersPerThread) * outputsX);
    if (filtersPerThread == 1) {
      if (checkCaseBounds) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalProbPool<Pooler), 4, 32, 4, 1, true>, hipFuncCachePreferL1);
        kLocalProbPool<Pooler, 4, 32, 4, 1, true><<<blocks, threads>>>(images->data_device, rnd->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, pooler);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalProbPool<Pooler), 4, 32, 4, 1, false>, hipFuncCachePreferL1);
        kLocalProbPool<Pooler, 4, 32, 4, 1, false><<<blocks, threads>>>(images->data_device, rnd->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, pooler);
      }
    } else {
      if (checkCaseBounds) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalProbPool<Pooler), 4, 32, 4, 2, true>, hipFuncCachePreferL1);
        kLocalProbPool<Pooler, 4, 32, 4, 2, true><<<blocks, threads>>>(images->data_device, rnd->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, pooler);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLocalProbPool<Pooler), 4, 32, 4, 2, false>, hipFuncCachePreferL1);
        kLocalProbPool<Pooler, 4, 32, 4, 2, false><<<blocks, threads>>>(images->data_device, rnd->data_device, target->data_device,
                                 imgSize, numFilters, numImages, subsX, startX, strideX, outputsX, pooler);
      }
    }

  }

  getLastCudaError("convLocalPool: kernel execution failed");
}

void convResponseNormUndo(cudamat* outGrads, cudamat* denoms, cudamat* inputs, cudamat* acts, cudamat* target, int numFilters,
             int sizeX, float addScale, float powScale, float scaleTargets, float scaleOutput) {
  int numImages = outGrads->size[0];
  int imgPixels = outGrads->size[1] / numFilters;

  int imgSize = int(sqrt(imgPixels));
  assert(imgSize * imgSize == imgPixels);

  assert(outGrads->size[1] == numFilters * imgPixels);
  
  //assert(denoms.isSameDims(outGrads));
  //assert(acts.isSameDims(denoms));
  //assert(!denoms.isTrans());
  //assert(!outGrads.isTrans());
  //assert(!acts.isTrans());
  //assert(!target.isTrans());
  //assert(outGrads.isContiguous());
  
  assert(numFilters % 16 == 0);
  
  //target.resize(outGrads);
  
  // First do acts := -2 x scale x acts x outGrads / denoms
  // so that the main routine only has to do an addition in its inner loop.
  int prelimEltsPerThread = 4;
  dim3 threads(128);
  dim3 blocks(MIN(512, DIVUP(outGrads->size[0]*outGrads->size[1],(threads.x * prelimEltsPerThread))));
  kRNormUndoPrelims<128, 4><<<blocks, threads>>>(acts->data_device, denoms->data_device, outGrads->data_device, outGrads->size[0]*outGrads->size[1], -2*addScale*powScale);
  
  // Now the main routine
  if (sizeX >= 6 && numFilters % 4 == 0) {
    // This one is faster for large regions (my tests show regions >= 6...)
    int imgsPerThread = 8;
    int filtersPerThread = 4;
    int bx = 16;
    bool checkCaseBounds = numImages % (bx*imgsPerThread) != 0;
    assert((imgsPerThread * bx) % 32 == 0);

    threads = dim3(bx, 16);
    blocks = dim3(DIVUP(imgSize, 4) * DIVUP(numImages, bx*imgsPerThread), DIVUP(imgSize, 4) * numFilters / filtersPerThread);
    if (checkCaseBounds) {
      if (scaleTargets == 0 && scaleOutput == 1) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kRNormUndo2<16), 8, 4, true, true>, hipFuncCachePreferL1);
        kRNormUndo2<16, 8, 4, true, true><<<blocks, threads>>>(outGrads->data_device, denoms->data_device, inputs->data_device, acts->data_device,
                                       target->data_device, imgSize, numFilters, numImages, sizeX, powScale,
                                       scaleTargets, scaleOutput);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kRNormUndo2<16), 8, 4, false, true>, hipFuncCachePreferL1);
        kRNormUndo2<16, 8, 4, false, true><<<blocks, threads>>>(outGrads->data_device, denoms->data_device, inputs->data_device, acts->data_device,
                                       target->data_device, imgSize, numFilters, numImages, sizeX, powScale,
                                       scaleTargets, scaleOutput);
      }
    } else {
      if (scaleTargets == 0 && scaleOutput == 1) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kRNormUndo2<16), 8, 4, true, false>, hipFuncCachePreferL1);
        kRNormUndo2<16, 8, 4, true, false><<<blocks, threads>>>(outGrads->data_device, denoms->data_device, inputs->data_device, acts->data_device,
                                       target->data_device, imgSize, numFilters, numImages, sizeX, powScale,
                                       scaleTargets, scaleOutput);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kRNormUndo2<16), 8, 4, false, false>, hipFuncCachePreferL1);
        kRNormUndo2<16, 8, 4, false, false><<<blocks, threads>>>(outGrads->data_device, denoms->data_device, inputs->data_device, acts->data_device,
                                       target->data_device, imgSize, numFilters, numImages, sizeX, powScale,
                                       scaleTargets, scaleOutput);
      }
    }
  } else {
    bool checkCaseBounds = numImages % 128 != 0;
    threads = dim3(32, 4);
    blocks = dim3(DIVUP(numImages,32*2) * imgSize, (numFilters / (4 * 2)) * imgSize);
    if (checkCaseBounds) { 
      if (scaleTargets == 0 && scaleOutput == 1) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kRNormUndo<4), 32, 2, 2, false, true>, hipFuncCachePreferL1);
        kRNormUndo<4, 32, 2, 2, false, true><<<blocks, threads>>>(outGrads->data_device, denoms->data_device, inputs->data_device, acts->data_device,
                                     target->data_device, imgSize, numFilters, numImages, sizeX, powScale,
                                     scaleTargets, scaleOutput);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kRNormUndo<4), 32, 2, 2, true, true>, hipFuncCachePreferL1);
        kRNormUndo<4, 32, 2, 2, true, true><<<blocks, threads>>>(outGrads->data_device, denoms->data_device, inputs->data_device, acts->data_device,
                                     target->data_device, imgSize, numFilters, numImages, sizeX, powScale,
                                     scaleTargets, scaleOutput);
      }
    } else {
      if (scaleTargets == 0 && scaleOutput == 1) {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kRNormUndo<4), 32, 2, 2, false, false>, hipFuncCachePreferL1);
        kRNormUndo<4, 32, 2, 2, false, false><<<blocks, threads>>>(outGrads->data_device, denoms->data_device, inputs->data_device, acts->data_device,
                                     target->data_device, imgSize, numFilters, numImages, sizeX, powScale,
                                     scaleTargets, scaleOutput);
      } else {
        hipFuncSetCacheConfig(reinterpret_cast<const void*>(kRNormUndo<4), 32, 2, 2, true, false>, hipFuncCachePreferL1);
        kRNormUndo<4, 32, 2, 2, true, false><<<blocks, threads>>>(outGrads->data_device, denoms->data_device, inputs->data_device, acts->data_device,
                                     target->data_device, imgSize, numFilters, numImages, sizeX, powScale,
                                     scaleTargets, scaleOutput);
      }
    }
  }
  getLastCudaError("kRNormUndo: kernel execution failed");
}






void convContrastNormCrossMap(cudamat* images, cudamat* meanDiffs, cudamat* denoms, cudamat* target,
                             int numFilters, int sizeF, float addScale, float powScale, float minDiv, bool blocked) {

    // printf("reza2:%d\n",images->size[0]);

    int numImages = images->size[0];
    int imgPixels = images->size[1] / numFilters;
    // printf("%d-%d-%d\n",images->size[1],numFilters,imgPixels);
    assert(images->size[1] == numFilters * imgPixels);
    int imgSize = int(sqrt(imgPixels));
    // printf("imgSize:%d-numFilters:%d-numImages:%d-sizeF:%d-addScale:%f-powScale:%f\n",imgSize,numFilters,numImages,sizeF,addScale,powScale);


    // int numImages = images.getNumCols();
    // int imgPixels = images.getNumRows() / numFilters;
    // assert(images.getNumRows() == numFilters * imgPixels);
    // int imgSize = int(sqrt(imgPixels));
    // assert(imgSize * imgSize == imgPixels);
    // assert(meanDiffs.isSameDims(images));
    // assert(sizeF > 0 && sizeF <= numFilters);
    
    // assert(!meanDiffs.isTrans());
    // assert(!images.isTrans());
    // assert(images.isContiguous());
    // assert(meanDiffs.isContiguous());
    // assert(numFilters % 16 == 0);

    // target.resize(images);
    // denoms.resize(images);
    // assert(target.isContiguous());

    bool checkCaseBounds = numImages % 128 != 0;
        
    dim3 threads(32, 4);
    dim3 blocks(DIVUP(numImages,32*4) * imgSize, (numFilters / 4) * imgSize);
    if (blocked) {
        if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(kFCNorm<4), 32, 4, true, true>, hipFuncCachePreferL1);
            kFCNorm<4, 32, 4, true, true><<<blocks, threads>>>(images->data_device, meanDiffs->data_device, denoms->data_device, target->data_device,
                                                                imgSize, numFilters, numImages, sizeF, addScale, powScale, minDiv);
        } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(kFCNorm<4), 32, 4, false, true>, hipFuncCachePreferL1);
            kFCNorm<4, 32, 4, false, true><<<blocks, threads>>>(images->data_device, meanDiffs->data_device, denoms->data_device, target->data_device,
                                                                imgSize, numFilters, numImages, sizeF, addScale, powScale, minDiv);
        }
    } else {
    if (checkCaseBounds) {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(kFCNorm<4), 32, 4, true, false>, hipFuncCachePreferL1);
            kFCNorm<4, 32, 4, true, false><<<blocks, threads>>>(images->data_device, meanDiffs->data_device, denoms->data_device, target->data_device,
                                                                imgSize, numFilters, numImages, sizeF, addScale, powScale, minDiv);
        } else {
            hipFuncSetCacheConfig(reinterpret_cast<const void*>(kFCNorm<4), 32, 4, false, false>, hipFuncCachePreferL1);
            kFCNorm<4, 32, 4, false, false><<<blocks, threads>>>(images->data_device, meanDiffs->data_device, denoms->data_device, target->data_device,
                                                                imgSize, numFilters, numImages, sizeF, addScale, powScale, minDiv);
        }
    }

    getLastCudaError("convContrastNormCrossMap: kernel execution failed");
}
















void convContrastNormUndo(cudamat* outGrads, cudamat* denoms,
                          cudamat* meanDiffs, cudamat* acts, cudamat* target,
                          int numFilters, int sizeX, float addScale,
                          float powScale, float scaleTargets, float scaleOutput) {
  convResponseNormUndo(outGrads, denoms, meanDiffs, acts, target, numFilters,
                       sizeX, addScale, powScale, scaleTargets, scaleOutput);
}

void convResponseNorm(cudamat* images, cudamat* denoms, cudamat* target, int numFilters, int sizeX, float addScale, float powScale) {
  convContrastNorm(images, images, denoms, target, numFilters, sizeX, addScale, powScale);
}


extern "C" {
// Convolutions.
extern void convUp(cudamat* images, cudamat* filters, cudamat* targets, int numModulesX, int paddingStart, int moduleStride, int numImgColors, int numGroups){
 // struct timeval start_tv, end_tv;
 // time_t sec;
 // time_t ms;
 // time_t diff;
 // gettimeofday(&start_tv, NULL);  


 filterActs(images, filters, targets, numModulesX, paddingStart, moduleStride, numImgColors, numGroups, 0, 1, true);

  // hipDeviceSynchronize();
  // gettimeofday(&end_tv, NULL);
  // sec = end_tv.tv_sec - start_tv.tv_sec;
  // ms = end_tv.tv_usec - start_tv.tv_usec;
  // diff = sec * 1000000 + ms;
  // printf("filterActs:%d--\t\t%fms\n",numModulesX , (double)((double)diff/1000.0));  
}
extern void convDown(cudamat* images, cudamat* filters, cudamat* targets, int imgSize, int paddingStart, int moduleStride, int numImgColors, int numGroups){
 imgActs(images, filters, targets, imgSize, paddingStart, moduleStride, numImgColors, numGroups, 0, 1, true);
}
extern void convOutp(cudamat* images, cudamat* hidSums, cudamat* targets, int numModulesX, int filterSize, int paddingStart, int moduleStride, int numImgColors, int numGroups, int partialSum){

// hipEvent_t start, stop; 
// float elapsedTime;
// hipEventCreate(&start); 
// hipEventCreate(&stop); 
// hipEventRecord(start,0);

weightActs(images, hidSums, targets, numModulesX, filterSize, paddingStart, moduleStride, numImgColors, numGroups, partialSum, 0, 1);

// hipEventRecord(stop,0); 
// hipEventSynchronize(stop);
// hipEventElapsedTime(&elapsedTime, start,stop); 
// printf("weightActs:%d---%f\n",numModulesX,elapsedTime);

}

// Local Connections.
extern void localUp(cudamat* images, cudamat* filters, cudamat* targets, int numModulesX, int paddingStart, int moduleStride, int numImgColors, int numGroups){


 filterActs(images, filters, targets, numModulesX, paddingStart, moduleStride, numImgColors, numGroups, 0, 1, false);

   
}
extern void localDown(cudamat* images, cudamat* filters, cudamat* targets, int imgSize, int paddingStart, int moduleStride, int numImgColors, int numGroups){
 imgActs(images, filters, targets, imgSize, paddingStart, moduleStride, numImgColors, numGroups, 0, 1, false);
}
extern void localOutp(cudamat* images, cudamat* hidSums, cudamat* targets, int numModulesX, int filterSize, int paddingStart, int moduleStride, int numImgColors, int numGroups, int partialSum){
 weightActs(images, hidSums, targets, numModulesX, filterSize, paddingStart, moduleStride, numImgColors, numGroups, 1, 0, 1);
}

// Response Normalization.
extern void ResponseNorm(cudamat* images, cudamat* denoms, cudamat* targets, int numFilters, int sizeX, float addScale, float powScale){
  convResponseNorm(images, denoms, targets, numFilters, sizeX, addScale, powScale);
}

extern void ResponseNormUndo(cudamat* outGrads, cudamat* denoms, cudamat* inputs, cudamat* acts, cudamat* targets, int numFilters, int sizeF, float addScale, float powScale){
  convResponseNormUndo(outGrads, denoms, inputs, acts, targets, numFilters, sizeF, addScale, powScale, 0, 1);
  // convResponseNormUndo(outGrads, denoms, inputs, acts, targets, numFilters,
                         // sizeF, addScale, powScale, 1, 0, 1);  
}
// Contrast Normalization.
extern void ContrastNorm(cudamat* images, cudamat* meanDiffs, cudamat* denoms, cudamat* targets, int numFilters, int sizeX, float addScale, float powScale){
  convContrastNorm(images, meanDiffs, denoms, targets, numFilters, sizeX, addScale, powScale);
}
extern void ContrastNormUndo(cudamat* outGrads, cudamat* denoms, cudamat* meanDiffs, cudamat* acts, cudamat* targets, int numFilters, int sizeX, float addScale, float powScale){
  convContrastNormUndo(outGrads, denoms, meanDiffs, acts, targets, numFilters, sizeX, addScale, powScale, 0, 1);
}

// Pooling.
extern void MaxPool(cudamat* images, cudamat* targets, int numFilters, int subsX,	int startX,	int strideX, int outputsX){
  MaxPooler mpooler;
  convLocalPool<MaxPooler>(images, targets, numFilters, subsX, startX, strideX, outputsX, mpooler);
}

//Added by Alireza
extern void AvgPool(cudamat* images, cudamat* targets, int numFilters, int subsX, int startX, int strideX, int outputsX){
  // AvgPooler mpooler(images->size[0] * images->size[1]);
  AvgPooler mpooler(subsX*subsX);
  convLocalPool<AvgPooler>(images, targets, numFilters, subsX, startX, strideX, outputsX, mpooler);
}

extern void ProbMaxPool(cudamat* images, cudamat* rnd, cudamat* targets, int numFilters, int subsX,	int startX,	int strideX, int outputsX){
  ProbMaxPooler mpooler;
  convLocalProbPool<ProbMaxPooler>(images, rnd, targets, numFilters, subsX, startX, strideX, outputsX, mpooler);
}

extern void MaxPoolUndo(cudamat* images, cudamat* maxGrads, cudamat* maxActs, cudamat* targets, int subsX, int startX, int strideX, int outputsX){
  convLocalMaxUndo(images, maxGrads, maxActs, targets, subsX, startX, strideX, outputsX, 0, 1);
}

extern void AvgPoolUndo(cudamat* avgGrads, cudamat* targets, int subsX, int startX, int strideX, int outputsX){
  convLocalAvgUndo(avgGrads, targets, subsX, startX, strideX, outputsX, 0, 1);
}




extern void convResponseNormCrossMap(cudamat* images, cudamat* denoms, cudamat* target, int numFilters, int sizeF, float addScale, float powScale, float minDiv, bool blocked) {
    // printf("reza:%d\n",images->size[0]);
    convContrastNormCrossMap(images, images, denoms, target, numFilters, sizeF, addScale, powScale, minDiv, blocked);
}


}
